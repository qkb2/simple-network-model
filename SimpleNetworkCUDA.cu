#include "hip/hip_runtime.h"
#include "SimpleNetworkCUDA.cuh"

__global__ void initialize_kernel(double* d_weights, int size, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state); // Initialize random number generator
        d_weights[idx] = hiprand_uniform(&state); // Generate random number between 0 and 1
    }
}

std::vector<double> SimpleNetworkCUDA::forward(const std::vector<double>& input, const SimpleMatrix<double>& layer)
{
    std::vector<double> output(layer.rows);

    // calculate product
#pragma omp parallel for
    for (int i = 0; i < layer.rows; i++)
    {
        double res = 0;
        for (int j = 0; j < layer.cols; j++)
        {
            res += input[j] * layer.matrix[i][j];
        }
        output[i] = res;
    }

    return output;
}

std::vector<double> SimpleNetworkCUDA::ReLU(const std::vector<double>& input)
{
    std::vector<double> output(input.size());

    // calculate Leaky ReLU for every value in vector
#pragma omp parallel for
    for (int i = 0; i < input.size(); i++)
    {
        double x = input[i];
        if (x > 0) {
            output[i] = x;
        }
        else
            output[i] = 0.01 * x;
    }
    return output;
}

void SimpleNetworkCUDA::randomize_weights()
{
    std::vector<size_t> sizes(layers.size());
    size_t sum_size = 0;
    for (size_t layer_it = 0; layer_it < layers.size(); layer_it++) {
        auto& layer = layers[layer_it];

        sizes[layer_it] = layer.rows * layer.cols * sizeof(double);
        sum_size += sizes[layer_it];
    }
    double* d_weights;

    hipMalloc(&d_weights, sum_size);

    double* res = (double*)malloc(sum_size);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (sum_size + threadsPerBlock - 1) / threadsPerBlock;

    initialize_kernel <<< threadsPerBlock, blocksPerGrid >>> (d_weights, sum_size, time(NULL));
    auto err = hipGetLastError();

    //hipDeviceSynchronize();

    // Copy data back to host
    hipMemcpy(res, d_weights, sum_size, hipMemcpyDeviceToHost);


    hipFree(d_weights);

    size_t elements = 0;
    for (int ij = 0; ij < layers.size(); ij++) 
    {
        auto& layer = layers[ij];
        for (int i = 0; i < layer.rows; ++i) {
            for (int j = 0; j < layer.cols; ++j) {
                // Calculate the index in the 1D array
                int index = i * layer.cols + j;
                // Assign the value from the 1D array to the matrix
                layer.matrix[i][j] = res[index + elements];
            }
        }
        elements += layer.cols * layer.rows;
    }
    free(res);
}

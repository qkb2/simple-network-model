﻿#include <iostream>
#include "SimpleNetwork.h"

int main() 
{
	SimpleNetwork* network = new SimpleNetwork(10);
	network->add_layer(100);
	network->add_layer(200);
	network->add_layer(100);
	network->add_layer(10);
	network->randomize_weights();

	std::vector<double> input = {0.1, 0.2, 0.3, 0.4, 0.5, 0.4, 0.3, 0.2, 0.1, 0.0};
	network->set_input(input);
	auto output = network->get_output();
	for (int i = 0; i < output.size(); i++)
	{
		std::cout << output[i] << " ";
	}

	return 0;
}
﻿#include <iostream>
#include <string>

#include "SimpleNetwork.h"
#include "SimpleNetworkMP.h"
#include "SimpleNetworkCUDA.cuh"

int main(int argc, char* argv[]) 
{
	if (argc < 3) return -1;
	std::string opt = argv[1];
	long input_size = atol(argv[2]);
	SimpleNetwork<double>* network;
	
	if (opt.compare("MP") == 0) {
		network = new SimpleNetworkMP<double>(input_size);
	}
	else if (opt.compare("CUDA") == 0) {
		network = new SimpleNetworkCUDA(input_size);
	}
	else
		network = new SimpleNetwork<double>(input_size);

	for (int i = 3; i < argc; i++)
	{
		long layer_size = atol(argv[i]);
		network->add_layer(layer_size);
	}

	network->randomize_weights();

	std::random_device dev;
	std::mt19937 rng(dev());
	std::uniform_real_distribution<> dist(-10.0, 10.0);
	std::vector<double> input;
	input.resize(input_size, 0);
	for (long i = 0; i < input_size; i++)
	{
		input[i] = (double) dist(rng);
	}
	network->set_input(input);
	
	auto output = network->get_output();
	for (int i = 0; i < output.size(); i++)
	{
		std::cout << output[i] << " ";
	}

	return 0;
}